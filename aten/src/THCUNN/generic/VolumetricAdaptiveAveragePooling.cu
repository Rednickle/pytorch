
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricAdaptiveAveragePooling.cu"
#else

#include "../common.h"

// 5d tensor B x D x T x H x W

void THNN_(VolumetricAdaptiveAveragePooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           int osizeT,
           int osizeW,
           int osizeH)
{
  THCUNN_assertSameGPU(state, 2, input, output);

  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 4 || input->dim() == 5), 2, input,
                  "non-empty 4D or 5D (batch mode) tensor expected for input, but got: %s");


  real *output_data;
  real *input_data;

  int64_t sizeD, isizeT, isizeH, isizeW;
  int64_t istrideD, istrideT, istrideH, istrideW;
  int64_t totalZ;

  if (input->dim() == 4) {
    sizeD = THTensor_sizeLegacyNoScalars(input, 0);
    isizeT = THTensor_sizeLegacyNoScalars(input, 1);
    isizeH = THTensor_sizeLegacyNoScalars(input, 2);
    isizeW = THTensor_sizeLegacyNoScalars(input, 3);

    istrideD = THTensor_strideLegacyNoScalars(input, 0);
    istrideT = THTensor_strideLegacyNoScalars(input, 1);
    istrideH = THTensor_strideLegacyNoScalars(input, 2);
    istrideW = THTensor_strideLegacyNoScalars(input, 3);

    THCTensor_(resize4d)(state, output, sizeD, osizeT, osizeH, osizeW);

    totalZ = sizeD * osizeT;
  } else {
    input = THCTensor_(newContiguous)(state, input);

    int64_t sizeB = THTensor_sizeLegacyNoScalars(input, 0);
    sizeD = THTensor_sizeLegacyNoScalars(input, 1);
    isizeT = THTensor_sizeLegacyNoScalars(input, 2);
    isizeH = THTensor_sizeLegacyNoScalars(input, 3);
    isizeW = THTensor_sizeLegacyNoScalars(input, 4);

    istrideD = THTensor_strideLegacyNoScalars(input, 1);
    istrideT = THTensor_strideLegacyNoScalars(input, 2);
    istrideH = THTensor_strideLegacyNoScalars(input, 3);
    istrideW = THTensor_strideLegacyNoScalars(input, 4);

    THCTensor_(resize5d)(state, output, sizeB, sizeD, osizeT, osizeH, osizeW);

    totalZ = sizeB * sizeD * osizeT;
  }

  input_data = THCTensor_(data)(state, input);
  output_data = THCTensor_(data)(state, output);

  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  // each H*W plane is processed by blocksH thread blocks
  int blocksH = max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);
    cunn_VolumetricAdaptiveAveragePooling_updateOutput_kernel
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        input_data, output_data, isizeT, isizeH, isizeW, osizeT, osizeH, osizeW,
        istrideD, istrideT, istrideH, istrideW, offsetZ
      );

    totalZ -= 65535;
    offsetZ += 65535;
    THCudaCheck(cudaGetLastError());
  }

  if (input->dim() == 5) {
    // clean
    THCTensor_(free)(state, input);
  }
}

void THNN_(VolumetricAdaptiveAveragePooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput)
{
  THCUNN_assertSameGPU(state, 3, input, gradOutput, gradInput);

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  real *gradInput_data;
  real *gradOutput_data;

  int64_t sizeD, isizeT, isizeH, isizeW;
  int64_t osizeT, osizeH, osizeW;
  int64_t totalZ;

  if (input->dim() == 4) {
    sizeD = THTensor_sizeLegacyNoScalars(input, 0);
    isizeT = THTensor_sizeLegacyNoScalars(input, 1);
    isizeH = THTensor_sizeLegacyNoScalars(input, 2);
    isizeW = THTensor_sizeLegacyNoScalars(input, 3);

    osizeT = THTensor_sizeLegacyNoScalars(gradOutput, 1);
    osizeH = THTensor_sizeLegacyNoScalars(gradOutput, 2);
    osizeW = THTensor_sizeLegacyNoScalars(gradOutput, 3);
  } else {
    sizeD = THTensor_sizeLegacyNoScalars(input, 1);
    isizeT = THTensor_sizeLegacyNoScalars(input, 2);
    isizeH = THTensor_sizeLegacyNoScalars(input, 3);
    isizeW = THTensor_sizeLegacyNoScalars(input, 4);

    osizeT = THTensor_sizeLegacyNoScalars(gradOutput, 2);
    osizeH = THTensor_sizeLegacyNoScalars(gradOutput, 3);
    osizeW = THTensor_sizeLegacyNoScalars(gradOutput, 4);
  }

  // somehow nonatomic is passing all test for volumetric case.
  bool atomic = false; //(isizeW%osizeW != 0) || (isizeH%osizeH != 0) || (isizeT%osizeT != 0);

  if (input->dim() == 4) {
    totalZ = atomic ? sizeD * osizeT : sizeD * isizeT;
  } else {
    int sizeB = THTensor_sizeLegacyNoScalars(input, 0);
    totalZ = atomic ? sizeB * sizeD * osizeT : sizeB * sizeD * isizeT;
  }

  gradInput_data = THCTensor_(data)(state, gradInput);
  gradOutput_data = THCTensor_(data)(state, gradOutput);

  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  // each H*W plane is processed by blocksH thread blocks
  int blocksH = max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);

    if (atomic)
    {
      cunn_atomic_VolumetricAdaptiveAveragePooling_updateGradInput_kernel
        <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
          gradInput_data, gradOutput_data, isizeT, isizeH, isizeW,
          osizeT, osizeH, osizeW, offsetZ
        );
    } else {
        cunn_VolumetricAdaptiveAveragePooling_updateGradInput_kernel
          <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
            gradInput_data, gradOutput_data, isizeT, isizeH, isizeW,
            osizeT, osizeH, osizeW, offsetZ
          );
    }

    totalZ -= 65535;
    offsetZ += 65535;
    THCudaCheck(cudaGetLastError());
  }
  // clean
  THCTensor_(free)(state, gradOutput);

}

#endif
