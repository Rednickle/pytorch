
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricAdaptiveMaxPooling.cu"
#else

#include "../common.h"

// 5d tensor B x D x T x H x W

void THNN_(VolumetricAdaptiveMaxPooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int osizeT,
           int osizeW,
           int osizeH)
{
  THCUNN_assertSameGPU(state, 3, input, output, indices);

  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 4 || input->dim() == 5), 2, input,
                  "4D or 5D (batch mode) tensor expected for input, but got: %s");

  THCIndex_t *indices_data;
  real *output_data;
  real *input_data;

  int64_t sizeD, isizeT, isizeH, isizeW;
  int64_t istrideD, istrideT, istrideH, istrideW;
  int64_t totalZ;

  if (input->dim() == 4) {
    sizeD = THTensor_sizeLegacyNoScalars(input, 0);
    isizeT = THTensor_sizeLegacyNoScalars(input, 1);
    isizeH = THTensor_sizeLegacyNoScalars(input, 2);
    isizeW = THTensor_sizeLegacyNoScalars(input, 3);

    istrideD = THTensor_strideLegacyNoScalars(input, 0);
    istrideT = THTensor_strideLegacyNoScalars(input, 1);
    istrideH = THTensor_strideLegacyNoScalars(input, 2);
    istrideW = THTensor_strideLegacyNoScalars(input, 3);

    THCTensor_(resize4d)(state, output, sizeD, osizeT, osizeH, osizeW);
    THCIndexTensor_(resize4d)(state, indices, sizeD, osizeT, osizeH, osizeW);

    totalZ = sizeD * osizeT;
  } else {
    input = THCTensor_(newContiguous)(state, input);

    int64_t sizeB = THTensor_sizeLegacyNoScalars(input, 0);
    sizeD = THTensor_sizeLegacyNoScalars(input, 1);
    isizeT = THTensor_sizeLegacyNoScalars(input, 2);
    isizeH = THTensor_sizeLegacyNoScalars(input, 3);
    isizeW = THTensor_sizeLegacyNoScalars(input, 4);

    istrideD = THTensor_strideLegacyNoScalars(input, 1);
    istrideT = THTensor_strideLegacyNoScalars(input, 2);
    istrideH = THTensor_strideLegacyNoScalars(input, 3);
    istrideW = THTensor_strideLegacyNoScalars(input, 4);

    THCTensor_(resize5d)(state, output, sizeB, sizeD, osizeT, osizeH, osizeW);
    THCIndexTensor_(resize5d)(state, indices, sizeB, sizeD, osizeT, osizeH, osizeW);

    totalZ = sizeB * sizeD * osizeT;
  }

  input_data = THCTensor_(data)(state, input);
  output_data = THCTensor_(data)(state, output);
  indices_data = THCIndexTensor_(data)(state, indices);

  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  // each H*W plane is processed by blocksH thread blocks
  int blocksH = max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);
    cunn_VolumetricAdaptiveMaxPooling_updateOutput_kernel
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        input_data, output_data, indices_data, isizeT, isizeH, isizeW,
        osizeT, osizeH, osizeW, istrideD, istrideT, istrideH, istrideW, offsetZ
      );

    totalZ -= 65535;
    offsetZ += 65535;
    THCudaCheck(cudaGetLastError());
  }

  if (input->dim() == 5) {
    // clean
    THCTensor_(free)(state, input);
  }
}

void THNN_(VolumetricAdaptiveMaxPooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices)
{
  THCUNN_assertSameGPU(state, 4, input, indices, gradOutput, gradInput);

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  THCIndex_t *indices_data;
  real *gradInput_data;
  real *gradOutput_data;

  int64_t sizeD, isizeT, isizeH, isizeW;
  int64_t osizeT, osizeH, osizeW;
  int64_t totalZ;

  if (input->dim() == 4) {
    sizeD = THTensor_sizeLegacyNoScalars(input, 0);
    isizeT = THTensor_sizeLegacyNoScalars(input, 1);
    isizeH = THTensor_sizeLegacyNoScalars(input, 2);
    isizeW = THTensor_sizeLegacyNoScalars(input, 3);

    osizeT = THTensor_sizeLegacyNoScalars(gradOutput, 1);
    osizeH = THTensor_sizeLegacyNoScalars(gradOutput, 2);
    osizeW = THTensor_sizeLegacyNoScalars(gradOutput, 3);
  } else {
    sizeD = THTensor_sizeLegacyNoScalars(input, 1);
    isizeT = THTensor_sizeLegacyNoScalars(input, 2);
    isizeH = THTensor_sizeLegacyNoScalars(input, 3);
    isizeW = THTensor_sizeLegacyNoScalars(input, 4);

    osizeT = THTensor_sizeLegacyNoScalars(gradOutput, 2);
    osizeH = THTensor_sizeLegacyNoScalars(gradOutput, 3);
    osizeW = THTensor_sizeLegacyNoScalars(gradOutput, 4);
  }

  bool atomic = (isizeW%osizeW != 0) || (isizeH%osizeH != 0) || (isizeT%osizeT != 0);

  if (input->dim() == 4) {
    totalZ = sizeD * osizeT;
  } else {
    int sizeB = THTensor_sizeLegacyNoScalars(input, 0);
    totalZ = sizeB * sizeD * osizeT;
  }

  indices_data = THCIndexTensor_(data)(state, indices);
  gradInput_data = THCTensor_(data)(state, gradInput);
  gradOutput_data = THCTensor_(data)(state, gradOutput);

  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  // each H*W plane is processed by blocksH thread blocks
  int blocksH = max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);

    if (atomic)
    {
      cunn_atomic_VolumetricAdaptiveMaxPooling_updateGradInput_kernel
        <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
          gradInput_data, gradOutput_data, indices_data,
          isizeT, isizeH, isizeW, osizeT, osizeH, osizeW, offsetZ
        );
    } else {
      cunn_VolumetricAdaptiveMaxPooling_updateGradInput_kernel
        <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
          gradInput_data, gradOutput_data, indices_data,
          isizeT, isizeH, isizeW, osizeT, osizeH, osizeW, offsetZ
        );
    }

    totalZ -= 65535;
    offsetZ += 65535;
    THCudaCheck(cudaGetLastError());
  }
  // clean
  THCTensor_(free)(state, gradOutput);
}

#endif
