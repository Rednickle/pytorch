
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MultiLabelMarginCriterion.cu"
#else

// TODO: improve error messages
void THNN_(MultiLabelMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           THCTensor *istarget,
           int64_t reduction)
{
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  THCTensor_(resizeAs)(state, istarget, input);

  if(input->dim() == 1)
  {
    int dim = THTensor_sizeLegacyNoScalars(input, 0);
    THArgCheck(!target->is_empty() && (target->dim() == 1) && (THTensor_sizeLegacyNoScalars(target, 0) == dim), 3,
        "inconsistent target size");
    THCTensor_(resize1d)(state, output, 1);

    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<real, accreal>
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, dim,
        reduction == Reduction::ElementwiseMean
        );
    THCudaCheck(cudaGetLastError());
  }
  else if(input->dim() == 2)
  {
    int nframe = THTensor_sizeLegacyNoScalars(input, 0);
    int dim = THTensor_sizeLegacyNoScalars(input, 1);
    THArgCheck(!target->is_empty() && (target->dim() == 2) && (THTensor_sizeLegacyNoScalars(target, 0) == nframe)
               && (THTensor_sizeLegacyNoScalars(target, 1) == dim), 3, "inconsistent target size");

    dim3 blocks(THTensor_sizeLegacyNoScalars(input, 0));
    dim3 threads(MULTILABELMARGIN_THREADS);

    if (reduction != Reduction::None)
    {
      THCTensor *output_tmp = THCTensor_(newWithSize1d)(state, THTensor_sizeLegacyNoScalars(input, 0));
      THCTensor_(resize1d)(state, output, 1);

      cunn_MultiLabelMarginCriterion_updateOutput_kernel<real, accreal>
        <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
          THCTensor_(data)(state, output_tmp),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          THCTensor_(data)(state, istarget),
          nframe, dim,
          reduction == Reduction::ElementwiseMean
          );
      THCudaCheck(cudaGetLastError());
      THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(THCTensor_(sumall)(state, output_tmp)));
      THCTensor_(free)(state, output_tmp);
    }
    else
    {
    THCTensor_(resize1d)(state, output, THTensor_sizeLegacyNoScalars(input, 0));

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<real, accreal>
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        nframe, dim,
        false
        );
    THCudaCheck(cudaGetLastError());
    }
  }
  else
    AT_ERROR("non-empty vector or matrix expected, got size: ", input->sizes());

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
}

void THNN_(MultiLabelMarginCriterion_updateGradInput)(
            THCState *state,
            THCTensor *input,
            THCIndexTensor *target,
            THCTensor *gradOutput,
            THCTensor *gradInput,
            THCTensor *istarget,
            int64_t reduction)
{
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  if(gradInput->dim() == 1)
  {
    int dim = THTensor_sizeLegacyNoScalars(gradInput, 0);
    THArgCheck(!target->is_empty() && (target->dim() == 1) && (THTensor_sizeLegacyNoScalars(target, 0) == dim), 3,
               "inconsistent target size");
    THArgCheck(!istarget->is_empty() && (istarget->dim() == 1) && (THTensor_sizeLegacyNoScalars(istarget, 0) == dim), 3,
               "inconsistent isTarget size");
    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<real, accreal>
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, THTensor_sizeLegacyNoScalars(gradInput, 0),
        reduction == Reduction::ElementwiseMean,
        reduction != Reduction::None);

  }
  else if(gradInput->dim() == 2)
  {
    int nframe = THTensor_sizeLegacyNoScalars(gradInput, 0);
    int dim = THTensor_sizeLegacyNoScalars(gradInput, 1);
    THArgCheck(!target->is_empty() && (target->dim() == 2) && (THTensor_sizeLegacyNoScalars(target, 0) == nframe)
               && (THTensor_sizeLegacyNoScalars(target, 1) == dim), 3, "inconsistent target size");
    THArgCheck(!istarget->is_empty() && (istarget->dim() == 2) && (THTensor_sizeLegacyNoScalars(istarget, 0) == nframe)
               && (THTensor_sizeLegacyNoScalars(istarget, 1) == dim), 3, "inconsistent isTarget size");
    dim3 blocks(THTensor_sizeLegacyNoScalars(gradInput, 0));
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<real, accreal>
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        THTensor_sizeLegacyNoScalars(gradInput, 0), gradInput->size(1),
        reduction == Reduction::ElementwiseMean,
        reduction != Reduction::None);
  }
  else
    AT_ERROR("non-empty vector or matrix expected, got size: ", gradInput->sizes());

  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
  THCTensor_(free)(state, gradOutput);
}

#endif
