
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SpatialMaxUnpooling.cu"
#else

void THNN_(SpatialMaxUnpooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int owidth, int oheight)
{
  THCUNN_assertSameGPU(state, 3, input, output, indices);
  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 3 || input->dim() == 4), 2, input,
                  "non-empty 3D or 4D (batch mode) tensor expected for input, but got: %s");
  THCUNN_check_shape_indices(state, indices, input);

  int64_t nInputCols, nInputRows, nInputPlane, batchSize;

  if (input->dim() == 3) {
    nInputCols = THTensor_sizeLegacyNoScalars(input, 2);
    nInputRows = THTensor_sizeLegacyNoScalars(input, 1);
    nInputPlane = THTensor_sizeLegacyNoScalars(input, 0);
    batchSize = 1;
  }
  else
  {
    nInputCols = THTensor_sizeLegacyNoScalars(input, 3);
    nInputRows = THTensor_sizeLegacyNoScalars(input, 2);
    nInputPlane = THTensor_sizeLegacyNoScalars(input, 1);
    batchSize = THTensor_sizeLegacyNoScalars(input, 0);
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  THCTensor_(resize4d)(state, output, batchSize, nInputPlane, oheight, owidth);
  THCTensor_(zero)(state, output);

  int count = THCTensor_(nElement)(state, input);

  MaxUnpoolForward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, THCTensor_(data)(state, input), THCIndexTensor_(data)(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCTensor_(data)(state, output));
  THCudaCheck(cudaGetLastError());

  if(input->dim() == 3)
    THCTensor_(resize3d)(state, output, nInputPlane, oheight, owidth);

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, indices);
}

void THNN_(SpatialMaxUnpooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices,
           int owidth, int oheight)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, indices, gradInput);
  THCUNN_check_shape_indices(state, indices, input);

  int64_t nInputCols, nInputRows, nInputPlane, batchSize;
  int dimw = 2;
  int dimh = 1;

  if (input->dim() == 3) {
    nInputPlane = THTensor_sizeLegacyNoScalars(input, 0);
    batchSize = 1;
  }
  else
  {
    ++dimw;
    ++dimh;
    nInputPlane = THTensor_sizeLegacyNoScalars(input, 1);
    batchSize = THTensor_sizeLegacyNoScalars(input, 0);
  }
  nInputCols = THTensor_sizeLegacyNoScalars(input, dimw);
  nInputRows = THTensor_sizeLegacyNoScalars(input, dimh);

  if(owidth!=THTensor_sizeLegacyNoScalars(gradOutput, dimw) || oheight!=gradOutput->size(dimh)){
     THError("Inconsistent gradOutput size. oheight= %d, owidth= %d, gradOutput: %dx%d",
             oheight, owidth,THTensor_sizeLegacyNoScalars(gradOutput, dimh),gradOutput->size(dimw));
  }

  input = THCTensor_(newContiguous)(state, input);
  indices = THCIndexTensor_(newContiguous)(state, indices);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  int count = THCTensor_(nElement)(state, input);

  MaxUnpoolBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, THCTensor_(data)(state, gradOutput), THCIndexTensor_(data)(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, oheight, owidth, THCTensor_(data)(state, gradInput));
  THCudaCheck(cudaGetLastError());

  // clean
  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, indices);
  THCTensor_(free)(state, gradOutput);
}

#endif
